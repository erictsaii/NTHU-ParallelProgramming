
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <sys/time.h>

#define d_max 64
#define bc 32
#define br 32

void input(char *input_filename);
void output(char *output_filename);
__global__ void flash_attention(float *q, float *k, float *v, float *o, int d, int tc, int N);

int B, N, d;
float *Q, *K, *V, *O;

int main(int argc, char *argv[]) {
    input(argv[1]);

    int tr = N / br, tc = N / bc;
    float *d_q, *d_k, *d_v, *d_o;
    size_t BND = B * N * d * sizeof(float);

    // Q
    hipHostRegister(Q, BND, hipHostRegisterDefault);
    hipMalloc(&d_q, BND);
    hipMemcpy(d_q, Q, BND, hipMemcpyHostToDevice);
    // K
    hipHostRegister(K, BND, hipHostRegisterDefault);
    hipMalloc(&d_k, BND);
    hipMemcpy(d_k, K, BND, hipMemcpyHostToDevice);
    // V
    hipHostRegister(V, BND, hipHostRegisterDefault);
    hipMalloc(&d_v, BND);
    hipMemcpy(d_v, V, BND, hipMemcpyHostToDevice);
    // O
    // cudaHostRegister(O, BND, cudaHostRegisterDefault);
    hipMalloc(&d_o, BND);
    // cudaMemcpy(d_o, O, BND, cudaMemcpyHostToDevice);
    
    // grid size and block size
    dim3 grid_size(tr, B);
    dim3 block_size(32, 32); // 32 * 32 threads

    // kernel function
    flash_attention<<<grid_size, block_size>>>(d_q, d_k, d_v, d_o, d, tc, N);

    // copy the output to host
    hipMemcpy(O, d_o, BND, hipMemcpyDeviceToHost);

    output(argv[2]);

    return 0;
}

void input(char *input_filename) {
    FILE *file = fopen(input_filename, "rb");

    fread(&B, sizeof(int), 1, file);
    fread(&N, sizeof(int), 1, file);
    fread(&d, sizeof(int), 1, file);

    size_t BND = B * N * d * sizeof(float);
    Q = (float *)malloc(BND);
    K = (float *)malloc(BND);
    V = (float *)malloc(BND);
    O = (float *)malloc(BND);

    int Nd = N * d;
    int offset = 0;

    for (int i = 0; i < B; ++i) {
        fread(Q + offset, sizeof(float), Nd, file);
        fread(K + offset, sizeof(float), Nd, file);
        fread(V + offset, sizeof(float), Nd, file);
        offset += Nd;
    }

    memset(O, 0x00, B * N * d * sizeof(float));

    fclose(file);
}


__global__ void flash_attention(float *q, float *k, float *v, float *o, int d, int tc, int N) {
    __shared__ float kj[bc * d_max]; 
    __shared__ float vj[bc * d_max];
    __shared__ float qi[br * d_max];
    __shared__ float oi[br * d_max];
    __shared__ float li[br];
    __shared__ float li_new[br];
    __shared__ float pij[br * bc];

    float mij = 3.0;
    int qo_offset = blockIdx.y * N * d + blockIdx.x * br * d;
    float sqrt_d = 1.0 / sqrtf(d);
    float pv = 0.0F;
    float tmp;

    // load qi
    for (int i = 0; i < d; i += 32) {
        qi[threadIdx.y * d + threadIdx.x + i] = q[qo_offset + threadIdx.y * d + threadIdx.x + i];
    }
    
    // load oi
    for (int i = 0; i < d; i += 32) {
        oi[threadIdx.y * d + threadIdx.x + i] = 0.0;
    }
   
    // load li
    if (threadIdx.y == 0) {
        li[threadIdx.x] = 0.0;
    }


    // start for-loop
    for (int j = 0; j < tc; ++j) {
        // load kj 
        int kjvj_offset = blockIdx.y * N * d + j * bc * d;
        for (int i = 0; i < d; i += 32) {
            kj[threadIdx.y * d + threadIdx.x + i] = k[kjvj_offset + threadIdx.y * d + threadIdx.x + i];
        }

        // load vj
        for (int i = 0; i < d; i += 32) {
            vj[threadIdx.y * d + threadIdx.x + i] = v[kjvj_offset + threadIdx.y * d + threadIdx.x + i];
        }
        __syncthreads();
        
        // QKDotAndScalar
        tmp = 0.0F;
        for (int t = 0; t < d; t++) {
            tmp += qi[threadIdx.y * d + t] * kj[threadIdx.x * d + t];
        }
        tmp *= sqrt_d;

        // MinusMaxAndExp
        pij[threadIdx.y * bc + threadIdx.x] = expf(tmp - mij);
        __syncthreads();

        // RowSum
        if (threadIdx.y == 0) {
            tmp = 0.0F;
            for (int i = 0; i < bc; ++i) {
                tmp += pij[threadIdx.x * bc + i];
            }
        }

        // UpdateMiLiOi
        if (threadIdx.y == 0) {
            li_new[threadIdx.x] = li[threadIdx.x] + tmp;
        }
        __syncthreads();
        for (int i = 0; i < d; i += 32) {
            pv = 0.0F;
            for (int t = 0; t < bc; ++t) {
                pv += pij[threadIdx.y * bc + t] * vj[t * d + threadIdx.x + i];
            } 
            oi[threadIdx.y * d + threadIdx.x + i] = (li[threadIdx.y]  * oi[threadIdx.y * d + threadIdx.x + i] + pv) / li_new[threadIdx.y];
        }
        if (threadIdx.y == 0) {
            li[threadIdx.x] = li_new[threadIdx.x];
        }
    }
  
    // update o
    for (int i = 0; i < d; i += 32) {
        o[qo_offset + threadIdx.y * d + threadIdx.x + i] = oi[threadIdx.y * d + threadIdx.x + i];
    }
}

void output(char *output_filename) {
    FILE *file = fopen(output_filename, "wb");

    fwrite(O, sizeof(float), B * N * d, file);

    fclose(file);
}