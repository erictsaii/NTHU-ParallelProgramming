
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <sys/time.h>

#define d_max 64
#define bc 32
#define br 32

void input(char *input_filename);
void output(char *output_filename);
void launch_flash_attention(float *q, float *k, float *v, float *o);

__global__ void flash_attention(float *q, float *k, float *v, float *o, float *l, float *m, int d, int tc);

int B, N, d;
float *Q, *K, *V, *O;

int main(int argc, char *argv[]) {
    input(argv[1]);

    for (int i = 0; i < B; i++) {
        launch_flash_attention(
            Q + (i * N * d), 
            K + (i * N * d), 
            V + (i * N * d), 
            O + (i * N * d)
        );
    }

    output(argv[2]);

    return 0;
}

void input(char *input_filename) {
    FILE *file = fopen(input_filename, "rb");

    fread(&B, sizeof(int), 1, file);
    fread(&N, sizeof(int), 1, file);
    fread(&d, sizeof(int), 1, file);

    size_t BND = B * N * d * sizeof(float);
    Q = (float *)malloc(BND);
    K = (float *)malloc(BND);
    V = (float *)malloc(BND);
    O = (float *)malloc(BND);

    int Nd = N * d;
    int offset = 0;

    for (int i = 0; i < B; ++i) {
        fread(Q + offset, sizeof(float), Nd, file);
        fread(K + offset, sizeof(float), Nd, file);
        fread(V + offset, sizeof(float), Nd, file);
        offset += Nd;
    }

    memset(O, 0x00, B * N * d * sizeof(float));

    fclose(file);
}

void launch_flash_attention(float *q, float *k, float *v, float *o) {
    float *l = (float *)malloc(N * sizeof(float));
    float *m = (float *)malloc(N * sizeof(float));
    memset(l, 0x00, N * sizeof(float));
    for (int i = 0; i < N; i++) {
        m[i] = FLT_MIN;
    }

    int tr = N / br, tc = N / bc;
    float *d_q, *d_k, *d_v, *d_o, *d_l, *d_m;
    // Q
    hipMalloc(&d_q, N*d*sizeof(float));
    hipMemcpy(d_q, q, N*d*sizeof(float), hipMemcpyHostToDevice);
    // K
    hipMalloc(&d_k, N*d*sizeof(float));
    hipMemcpy(d_k, k, N*d*sizeof(float), hipMemcpyHostToDevice);
    // V
    hipMalloc(&d_v, N*d*sizeof(float));
    hipMemcpy(d_v, v, N*d*sizeof(float), hipMemcpyHostToDevice);
    // O
    hipMalloc(&d_o, N*d*sizeof(float));
    hipMemcpy(d_o, o, N*d*sizeof(float), hipMemcpyHostToDevice);
    // l
    hipMalloc(&d_l, N*sizeof(float));
    hipMemcpy(d_l, l, N*sizeof(float), hipMemcpyHostToDevice);
    // m
    hipMalloc(&d_m, N*sizeof(float));
    hipMemcpy(d_m, m, N*sizeof(float), hipMemcpyHostToDevice);
    
    // grid size and block size
    dim3 grid_size(tr);
    dim3 block_size(32, 32); // 32 * 32 threads

    // kernel function
    flash_attention<<<grid_size, block_size>>>(d_q, d_k, d_v, d_o, d_l, d_m, d, tc);

    // copy the output to host
    hipMemcpy(o, d_o, N*d*sizeof(float), hipMemcpyDeviceToHost);
}

__global__ void flash_attention(float *q, float *k, float *v, float *o, float *l, float *m, int d, int tc) {
    __shared__ float kj[bc * d_max]; 
    __shared__ float vj[bc * d_max];
    __shared__ float qi[br * d_max];
    __shared__ float oi[br * d_max];
    __shared__ float li[br];
    __shared__ float mi[br];
    __shared__ float li_new[br];
    __shared__ float mi_new[br];

    __shared__ float sij[br * bc];
    __shared__ float pij[br * bc];
    __shared__ float mij[br];
    __shared__ float lij[br];

    int qo_offset = blockIdx.x * br * d;
    int lm_offset = blockIdx.x * br;
    float sqrt_d = 1.0 / sqrtf(d);

    // load qi
    for (int i = 0; i < d; i += 32) {
        qi[threadIdx.y * d + threadIdx.x + i] = q[qo_offset + threadIdx.y * d + threadIdx.x + i];
    }
    
    // load oi
    for (int i = 0; i < d; i += 32) {
        oi[threadIdx.y * d + threadIdx.x + i] = o[qo_offset + threadIdx.y * d + threadIdx.x + i];
    }
   
    // load li
    if (threadIdx.y == 0) {
        li[threadIdx.x] = l[lm_offset + threadIdx.x];
    }
  
    // load mi
    if (threadIdx.y == 0){
        mi[threadIdx.x] = m[lm_offset + threadIdx.x];
    }
    __syncthreads();

    // start for-loop
    for (int j = 0; j < tc; ++j) {
        // load kj 
        int kjvj_offset = j * bc * d;
        for (int i = 0; i < d; i += 32) {
            kj[threadIdx.y * d + threadIdx.x + i] = k[kjvj_offset + threadIdx.y * d + threadIdx.x + i];
        }

        // load vj
        for (int i = 0; i < d; i += 32) {
            vj[threadIdx.y * d + threadIdx.x + i] = v[kjvj_offset + threadIdx.y * d + threadIdx.x + i];
        }
        __syncthreads();
        
        // QKDotAndScalar
        sij[threadIdx.y * bc + threadIdx.x] = 0.0F;
        for (int t = 0; t < d; t++) {
            sij[threadIdx.y * bc + threadIdx.x] += qi[threadIdx.y * d + t] * kj[threadIdx.x * d + t];
        }
        sij[threadIdx.y * bc + threadIdx.x] *= sqrt_d;
        __syncthreads();

        // RowMax (turn to threadIdx.y==0 ?)
        if (threadIdx.x == 0) {
            mij[threadIdx.y] = sij[threadIdx.y * bc];
            for (int i = 0; i < bc; ++i) {
                mij[threadIdx.y] = fmaxf(mij[threadIdx.y], sij[threadIdx.y * bc + i]);
            }
        }
        __syncthreads();

        // MinusMaxAndExp
        pij[threadIdx.y * bc + threadIdx.x] = expf(sij[threadIdx.y  * bc + threadIdx.x] - mij[threadIdx.y]);
        __syncthreads();

        // RowSum
        if (threadIdx.x == 0) {
            lij[threadIdx.y] = 0.0F;
            for (int i = 0; i < bc; ++i) {
                lij[threadIdx.y] += pij[threadIdx.y * bc + i];
            }
        }
        __syncthreads();

        // UpdateMiLiOi
        if (threadIdx.y == 0) {
            mi_new[threadIdx.x] = fmaxf(mi[threadIdx.x], mij[threadIdx.x]);
            li_new[threadIdx.x] = expf(mi[threadIdx.x] - mi_new[threadIdx.x]) * li[threadIdx.x] + expf(mij[threadIdx.x] - mi_new[threadIdx.x]) * lij[threadIdx.x];
        }
        __syncthreads();
        for (int i = 0; i < d; i += 32) {
            float pv = 0.0F;
            for (int t = 0; t < bc; ++t) {
                pv += pij[threadIdx.y * bc + t] * vj[t * d + threadIdx.x + i];
            } 
            oi[threadIdx.y * d + threadIdx.x + i] = (li[threadIdx.y] * expf(mi[threadIdx.y] - mi_new[threadIdx.y]) * oi[threadIdx.y * d + threadIdx.x + i] + expf(mij[threadIdx.y] - mi_new[threadIdx.y]) * pv) / li_new[threadIdx.y];
        }
        if (threadIdx.y == 0) {
            mi[threadIdx.x] = mi_new[threadIdx.x];
            li[threadIdx.x] = li_new[threadIdx.x];
        }
        __syncthreads();
    }
  
    // update o
    for (int i = 0; i < d; i += 32) {
        o[qo_offset + threadIdx.y * d + threadIdx.x + i] = oi[threadIdx.y * d + threadIdx.x + i];
    }
}

void output(char *output_filename) {
    FILE *file = fopen(output_filename, "wb");

    fwrite(O, sizeof(float), B * N * d, file);

    free(Q);
    free(K);
    free(V);
    free(O);

    fclose(file);
}